#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>

#define NUM_SIMULATIONS 1000000
#define NUM_TIMESTEPS 5000
#define DT 0.01

// Pendulum parameters
#define LENGTH 1.0
#define MASS 1.0
#define GRAVITY 3.0

// PID parameters
#define KP 3.0
#define KI 0.1
#define KD 0.5

__global__ void init(unsigned int seed, hiprandState_t* states) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= NUM_SIMULATIONS) return;

    hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void simulate(hiprandState_t* states, double *state, double *initial_state, double *final_state) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= NUM_SIMULATIONS) return;

    // Initial state
    double theta = hiprand_uniform_double(&states[idx]) * 2 * M_PI - M_PI; // -pi to pi
    double omega = hiprand_uniform_double(&states[idx]) * 2 - 1; // -1 to 1

    initial_state[idx*2] = theta;
    initial_state[idx*2+1] = omega;

    double integral = 0;
    double previous_error = 0;

    for (int t = 0; t < NUM_TIMESTEPS; t++) {
        // PID controller
        double error = 0 - theta; // error is difference from upright position
        integral += error * DT;
        double derivative = (error - previous_error) / DT;
        double control = KP * error + KI * integral + KD * derivative;
        previous_error = error;

        // Pendulum dynamics
        double alpha = GRAVITY/LENGTH * sin(theta) + control/MASS/LENGTH/LENGTH;

        // Update state using Euler integration
        theta += DT * omega;
        omega += DT * alpha;

        // Write new state back to global memory
        state[idx*2] = theta;
        state[idx*2+1] = omega;
    }
    
    final_state[idx*2] = theta;
    final_state[idx*2+1] = omega;
}

int main() {
    double *d_state;
    hiprandState_t *d_states;
    double *d_initial_state;
    double *d_final_state;

    hipMalloc((void**)&d_state, NUM_SIMULATIONS*2*sizeof(double));
    hipMalloc((void**)&d_states, NUM_SIMULATIONS*sizeof(hiprandState_t));
    hipMalloc((void**)&d_initial_state, NUM_SIMULATIONS*2*sizeof(double));
    hipMalloc((void**)&d_final_state, NUM_SIMULATIONS*2*sizeof(double));

    init<<<(NUM_SIMULATIONS + 255) / 256, 256>>>(time(NULL), d_states);

    simulate <<< (NUM_SIMULATIONS + 255) / 256, 256 >>> (d_states, d_state, d_initial_state, d_final_state);

    double *h_initial_state = new double[NUM_SIMULATIONS*2];
    double *h_final_state = new double[NUM_SIMULATIONS*2];

    hipMemcpy(h_initial_state, d_initial_state, NUM_SIMULATIONS*2*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_final_state, d_final_state, NUM_SIMULATIONS*2*sizeof(double), hipMemcpyDeviceToHost);

    // Write initial states to binary file
    std::ofstream initial_file("initial_states.bin", std::ios::binary);
    initial_file.write(reinterpret_cast<char*>(h_initial_state), NUM_SIMULATIONS*2*sizeof(double));
    initial_file.close();

    // Write final states to binary file
    std::ofstream final_file("final_states.bin", std::ios::binary);
    final_file.write(reinterpret_cast<char*>(h_final_state), NUM_SIMULATIONS*2*sizeof(double));
    final_file.close();

    delete[] h_initial_state;
    delete[] h_final_state;
    hipFree(d_state);
    hipFree(d_states);
    hipFree(d_initial_state);
    hipFree(d_final_state);
    return 0;
}


